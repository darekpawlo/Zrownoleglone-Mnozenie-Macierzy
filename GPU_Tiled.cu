#include "hip/hip_runtime.h"
// Tiled
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#define N 20000  // Rozmair macierzy NxN
#define TILE_WIDTH 16

__global__ void multiplyKernel(int* C, const int* A, const int* B, int n)
{
    // Bufory w pamięci współdzielonej dla kafelków macierzy A i B
    __shared__ int s_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ int s_B[TILE_WIDTH][TILE_WIDTH];

    // Obliczenie indeksów wiersza i kolumny dla każdego wątku
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    int value = 0;

    // Pętla po kafelkach
    for (int m = 0; m < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {

        // Wczytanie danych do pamięci współdzielonej z pamięci globalnej
        if (row < n && (m * TILE_WIDTH + threadIdx.x) < n)
            s_A[threadIdx.y][threadIdx.x] = A[row * n + m * TILE_WIDTH + threadIdx.x];
        else
            s_A[threadIdx.y][threadIdx.x] = 0;

        if (col < n && (m * TILE_WIDTH + threadIdx.y) < n)
            s_B[threadIdx.y][threadIdx.x] = B[(m * TILE_WIDTH + threadIdx.y) * n + col];
        else
            s_B[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        // Mnożenie kafelków

        for (int k = 0; k < TILE_WIDTH; ++k)
            value += s_A[threadIdx.y][k] * s_B[k][threadIdx.x];

        __syncthreads();
    }

    // Zapis wyniku do macierzy wynikowej C
    if (row < n && col < n)
        C[row * n + col] = value;
}
// Kernel do generowania losowych liczb dla macierzy a i b
__global__ void generateRandomMatrix(int* a, int* b, int size, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        // Generowanie liczb pseudolosowych w przedziale 1-10
        int randomValueA = (hiprand(&state) % 10) + 1;
        int randomValueB = (hiprand(&state) % 10) + 1;

        a[idx] = randomValueA;
        b[idx] = randomValueB;
    }
}
int main()
{
    const int arraySize = N;
    int* d_a;
    int* d_b;
    int* d_c;
    int matrixSize = arraySize * arraySize;
    int* a = (int*)malloc(arraySize * arraySize * sizeof(int));
    int* b = (int*)malloc(arraySize * arraySize * sizeof(int));
    int* c = (int*)malloc(arraySize * arraySize * sizeof(int));
    hipMalloc((void**)&d_a, matrixSize * sizeof(int));
    hipMalloc((void**)&d_b, matrixSize * sizeof(int));
    hipMalloc((void**)&d_c, matrixSize * sizeof(int));
    // Uruchomienie kernela do generowania losowych liczb
    int threadsPerBlock = 256;
    int blocksPerGrid = (matrixSize + threadsPerBlock - 1) / threadsPerBlock;
    generateRandomMatrix << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, matrixSize, time(0));
    // Synchronizacja
    hipDeviceSynchronize();

    // Kopiowanie danych z GPU do CPU
    hipMemcpy(a, d_a, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    auto start = std::chrono::high_resolution_clock::now();
    // Mnożenie macierzy na GPU
   // Konfiguracja siatki i bloków wątków
    dim3 threadsPerBlockk(TILE_WIDTH, TILE_WIDTH); // Max 32x32
    dim3 blocksPerGridd((N + threadsPerBlockk.x - 1) / threadsPerBlockk.x, (N + threadsPerBlockk.y - 1) / threadsPerBlockk.y);


    // Uruchomienie kernela na GPU
    multiplyKernel << <blocksPerGridd, threadsPerBlockk >> > (d_c, d_a, d_b, arraySize);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    hipMemcpy(c, d_c, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Czas wykonania tailed: " << duration.count() << " milliseconds" << std::endl;
    // Wyświetlenie 
   /* printf("Macierz A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", a[i * arraySize + j]);
        }
        printf("\n");
    }
    printf("Macierz B:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", b[i * arraySize + j]);
        }
        printf("\n");
    }
    printf("Macierz C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i * arraySize + j]);
        }
        printf("\n");
    }*/

    // Zwalnianie pamięci
    free(a);
    free(b);
    free(c);

    return 0;
}
