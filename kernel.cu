#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 4  // Rozmair macierzy NxN

hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void multiplyKernel(int* C, const int* A, const int* B, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int value = 0;
        for (int k = 0; k < n; ++k) {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}
// Kernel do generowania losowych liczb dla macierzy a i b
__global__ void generateRandomMatrix(int* a, int* b, int size, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        // Generowanie liczb pseudolosowych w przedziale 1-10
        int randomValueA = (hiprand(&state) % 10) + 1;
        int randomValueB = (hiprand(&state) % 10) + 1;

        a[idx] = randomValueA;
        b[idx] = randomValueB;
    }
}
int main()
{
    const int arraySize = N;
    int* d_a;
    int* d_b;
    int matrixSize = arraySize * arraySize;
    int* a = (int*)malloc(arraySize * arraySize * sizeof(int));
    int* b = (int*)malloc(arraySize * arraySize * sizeof(int));
    int* c = (int*)malloc(arraySize * arraySize * sizeof(int));
    hipMalloc((void**)&d_a, matrixSize * sizeof(int));
    hipMalloc((void**)&d_b, matrixSize * sizeof(int));
    // Uruchomienie kernela do generowania losowych liczb
    int threadsPerBlock = 256;
    int blocksPerGrid = (matrixSize + threadsPerBlock - 1) / threadsPerBlock;
    generateRandomMatrix << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, matrixSize, time(0));
    // Synchronizacja
    hipDeviceSynchronize();

    // Kopiowanie danych z GPU do CPU
    hipMemcpy(a, d_a, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    // Mnożenie macierzy na GPU
    hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }
    // Wyświetlenie 
    printf("Macierz A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", a[i * arraySize + j]);
        }
        printf("\n");
    }
    printf("Macierz B:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", b[i * arraySize + j]);
        }
        printf("\n");
    }
    printf("Macierz C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i * arraySize + j]);
        }
        printf("\n");
    }

    // Zwalnianie pamięci
    free(a);
    free(b);
    free(c);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Funkcja pomocnicza do mnożenia macierzy z użyciem CUDA
hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Wybór urządzenia CUDA
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Przydzielenie pamięci na GPU dla trzech macierzy (dwie wejściowe, jedna wynikowa)
    cudaStatus = hipMalloc((void**)&dev_c, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Kopiowanie macierzy wejściowych z hosta na urządzenie (CPU -> GPU)
    cudaStatus = hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Konfiguracja siatki i bloków wątków
    dim3 threadsPerBlock(32, 32); // Max 32x32
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);


    // Uruchomienie kernela na GPU
    multiplyKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);

    // Sprawdzenie, czy nie wystąpiły błędy podczas uruchamiania kernela
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Oczekiwanie na zakończenie wszystkich wątków
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", cudaStatus);
        goto Error;
    }

    // Kopiowanie wynikowej macierzy z GPU na hosta
    cudaStatus = hipMemcpy(c, dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
