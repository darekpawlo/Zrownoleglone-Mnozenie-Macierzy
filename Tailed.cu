#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 2000  // Rozmiar macierzy NxN
#define TILE_WIDTH 16

hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void tiledMultiplyKernel(int* C, const int* A, const int* B, int n)
{
    __shared__ int ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ int ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;   int by = blockIdx.y;
    int tx = threadIdx.x;  int ty = threadIdx.y;

    // Identyfikacja wiersza i kolumny elementu C do obliczenia
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    int Pvalue = 0;

    // Pętla po wszystkich kafelkach wymaganych do obliczenia elementu C
    for (int m = 0; m < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {

        // Współdzielone ładowanie kafelków do pamięci współdzielonej
        if (Row < n && m * TILE_WIDTH + tx < n)
            ds_A[ty][tx] = A[Row * n + m * TILE_WIDTH + tx];
        else
            ds_A[ty][tx] = 0;

        if (Col < n && m * TILE_WIDTH + ty < n)
            ds_B[ty][tx] = B[(m * TILE_WIDTH + ty) * n + Col];
        else
            ds_B[ty][tx] = 0;

        __syncthreads();

        // Mnożenie dwóch kafelków
        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_A[ty][k] * ds_B[k][tx];

        __syncthreads();
    }

    if (Row < n && Col < n)
        C[Row * n + Col] = Pvalue;
}

// Kernel do generowania losowych liczb dla macierzy a i b
__global__ void generateRandomMatrix(int* a, int* b, int size, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        // Generowanie liczb pseudolosowych w przedziale 1-10
        int randomValueA = (hiprand(&state) % 10) + 1;
        int randomValueB = (hiprand(&state) % 10) + 1;

        a[idx] = randomValueA;
        b[idx] = randomValueB;
    }
}

int main()
{
    const int arraySize = N;
    int* d_a;
    int* d_b;
    int matrixSize = arraySize * arraySize;
    int* a = (int*)malloc(matrixSize * sizeof(int));
    int* b = (int*)malloc(matrixSize * sizeof(int));
    int* c = (int*)malloc(matrixSize * sizeof(int));
    hipMalloc((void**)&d_a, matrixSize * sizeof(int));
    hipMalloc((void**)&d_b, matrixSize * sizeof(int));

    // Uruchomienie kernela do generowania losowych liczb
    int threadsPerBlock = 256;
    int blocksPerGrid = (matrixSize + threadsPerBlock - 1) / threadsPerBlock;
    generateRandomMatrix << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, matrixSize, time(0));

    // Synchronizacja
    hipDeviceSynchronize();

    // Kopiowanie danych z GPU do CPU
    hipMemcpy(a, d_a, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);

    // Mnożenie macierzy na GPU
    hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }

    // Wyświetlenie części macierzy (dla dużych N ograniczamy wyświetlanie)
    int displaySize = (N > 10) ? 10 : N;
    printf("Macierz A:\n");
    for (int i = 0; i < displaySize; i++) {
        for (int j = 0; j < displaySize; j++) {
            printf("%d ", a[i * arraySize + j]);
        }
        printf("\n");
    }
    if (N > 10) printf("...\n");

    printf("Macierz B:\n");
    for (int i = 0; i < displaySize; i++) {
        for (int j = 0; j < displaySize; j++) {
            printf("%d ", b[i * arraySize + j]);
        }
        printf("\n");
    }
    if (N > 10) printf("...\n");

    printf("Macierz C:\n");
    for (int i = 0; i < displaySize; i++) {
        for (int j = 0; j < displaySize; j++) {
            printf("%d ", c[i * arraySize + j]);
        }
        printf("\n");
    }
    if (N > 10) printf("...\n");

    // Zwalnianie pamięci
    free(a);
    free(b);
    free(c);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Funkcja pomocnicza do mnożenia macierzy z użyciem CUDA
hipError_t multiplyWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Wybór urządzenia CUDA
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Czy masz zainstalowaną kartę CUDA?");
        goto Error;
    }

    // Przydzielenie pamięci na GPU dla trzech macierzy
    cudaStatus = hipMalloc((void**)&dev_c, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Kopiowanie macierzy wejściowych z hosta na urządzenie (CPU -> GPU)
    cudaStatus = hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Konfiguracja siatki i bloków wątków
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 blocksPerGrid((size + TILE_WIDTH - 1) / TILE_WIDTH, (size + TILE_WIDTH - 1) / TILE_WIDTH);

    // Uruchomienie kernela na GPU
    tiledMultiplyKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);

    // Sprawdzenie błędów podczas uruchamiania kernela
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "tiledMultiplyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Oczekiwanie na zakończenie wszystkich wątków
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching tiledMultiplyKernel!\n", cudaStatus);
        goto Error;
    }

    // Kopiowanie wynikowej macierzy z GPU na hosta
    cudaStatus = hipMemcpy(c, dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
